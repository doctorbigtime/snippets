#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void mmult(double* A, double* B, double* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < N) && (col < N))
    {
        double dot = .0;
        for(int i = 0; i < N; ++i)
            dot += A[row*N + i] * B[col + i*N];
        C[row*N + col] = dot;
    }
}

#define CUDA_CHECK(stmt) do { \
    hipError_t err = stmt; \
    if(err != hipSuccess) { \
        std::cerr << "Failed to run statement: " << #stmt << std::endl; \
        return -1; \
    } \
} while(0)


extern bool do_cuda_mmult(double* A, double* B, double* C, int N)
{
    double *deviceA, *deviceB, *deviceC;
    int bytes = sizeof(double) * N * N;
    CUDA_CHECK(hipMalloc((void**)&deviceA, bytes));
    CUDA_CHECK(hipMalloc((void**)&deviceB, bytes));
    CUDA_CHECK(hipMalloc((void**)&deviceC, bytes));

    CUDA_CHECK(hipMemcpy(deviceA, A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceB, B, bytes, hipMemcpyHostToDevice));
    //hipMemcpy(deviceC, C, bytes, hipMemcpyHostToDevice);

    static unsigned int constexpr TILE_WIDTH = 8;
    dim3 dimGrid{ (N-1)/TILE_WIDTH+1, (N-1)/TILE_WIDTH+1, 1u };
    dim3 dimBlock{ TILE_WIDTH, TILE_WIDTH, 1 };

    mmult<<< dimGrid, dimBlock >>>(deviceA, deviceB, deviceC, N);

    hipDeviceSynchronize();
    CUDA_CHECK(hipMemcpy(C, deviceC, bytes, hipMemcpyDeviceToHost));
    return true;
}

